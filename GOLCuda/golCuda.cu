#include "hip/hip_runtime.h"
/***********************
Conway Game of Life
Cuda version
************************/

#include <stdio.h>
#include <stdlib.h>
#include <sys/stat.h>
#include <fcntl.h>
#include <unistd.h>
#include "timer.h"
#include <hip/hip_runtime.h>

#define DATAFILE "../TestFiles/960.txt"
#define DIM 960 /*Dimension of input file*/
#define ITERS 150
#define CUDA_BLOCK_DIM 16 /*Dimension of cuda-blocks (how many threads in each direction)*/

/*Kernel to fill columns of the extra perimeter*/
__global__ void fill_columns(char* array)
{
    /*Find unique thread id*/
    int thread_id = blockDim.x * blockIdx.x + threadIdx.x;

    if(thread_id < DIM+2)
    {
        array[thread_id*(DIM+2)+DIM+1] = array[thread_id*(DIM+2)+1];
        array[thread_id*(DIM+2)] = array[thread_id*(DIM+2) + DIM];    
    }
}

/*Kernel to fill rows of the extra perimeter*/
__global__ void fill_rows(char* array)
{
    /*Find unique thread id*/
    int thread_id = blockDim.x * blockIdx.x + threadIdx.x + 1;

    if (thread_id < DIM+1)
    {
        array[(DIM+2)*(DIM+1)+thread_id] = array[(DIM+2)+thread_id];
        array[thread_id] = array[(DIM+2)*DIM + thread_id];
    }
}

__global__ void step(char* prev,char* next/*,int* global_cell_sum,int* flag*/)
{

  /*Shared memory between threads in a block*/
  __shared__ int shared_arr[CUDA_BLOCK_DIM][CUDA_BLOCK_DIM]; /*shared memory between threads in a block*/

  /*Thread id corresponding to the global array*/
  int x = blockIdx.x * (blockDim.x -2) + threadIdx.x;
  int y = blockIdx.y * (blockDim.y -2) + threadIdx.y;
  int thread_id = x + y*(DIM+2);

  /*Thread id corresponding to the local array*/
  int i = threadIdx.y;
  int j = threadIdx.x;
  int neighbors;
  
  /*int cell_sum = 0;*/ /*Number of alive cells at the next generation*/

  /*Copy elements into shared memory*/
  if ((x < (DIM+2)) && (y < (DIM+2))) shared_arr[i][j] = prev[thread_id];

  /*Wait until all threads write to the shared memory*/
  __syncthreads();

  /*Check if the thread id's are out of bounds*/
  if ((x < (DIM+1)) && (y < (DIM+1)) && (i != 0) && (i != (blockDim.y-1)) && (j != 0) && (j != (blockDim.x-1))){
      /*Calculate the number of neighbors*/
      neighbors = (shared_arr[i+1][j]-'0') + (shared_arr[i-1][j]-'0') + (shared_arr[i][j+1]-'0') + (shared_arr[i][j-1]-'0') + 
      (shared_arr[i+1][j+1]-'0') + (shared_arr[i-1][j-1]-'0') + (shared_arr[i-1][j+1]-'0') + (shared_arr[i+1][j-1]-'0');
              
      if((shared_arr[i][j] == '0') && (neighbors == 3)){
        next[thread_id] = '1';
        /*cell_sum++;*/
      }
      else if(shared_arr[i][j] == '1'){
      	if(neighbors < 2){
 	  next[thread_id] = '0';
	  /**flag = 1;*/
	}	
        else if(neighbors < 4){
          next[thread_id] = '1';
          /*cell_sum++;*/
        }
        else{
 	  next[thread_id] = '0';
          /**flag = 1;*/
	}
      }
      else next[thread_id] = '0';
  }
  
  /*atomicAdd(global_cell_sum,cell_sum);*/
}


int main(int argc, char* argv[])
{
  int i/*,n*/;
  char* h_array;   /*host array*/
  char* dev_array1; /*previous generation device array*/
  char* dev_array2; /*next generation device array*/
  char* temp_arr;
  double start,finish;
  
  int fd = open(DATAFILE, O_RDONLY);
  if(fd < 0){
    fprintf(stderr, "Could not open file \"%s\"\n", DATAFILE);
    return -1;
  }
   
  h_array = (char*)malloc((DIM+2)*(DIM+2)*sizeof(char));
  
  /*Read the grid from the file into the array, skipping positions that correspond to the perimeter*/
  i = DIM+3;
  while(read(fd,&h_array[i],DIM)){
    i += DIM+2;
  }
  close(fd);
 
  /*struct stat st = {0};
  if (stat("./generations", &st) == -1)
    mkdir("./generations", 0700);*/

  /*Block size and grid size for threads to fill rows and columns*/
  dim3 rc_block_size(CUDA_BLOCK_DIM);
  dim3 rows_grid_size((int)ceil((DIM)/(float)rc_block_size.x));
  dim3 cols_grid_size((int)ceil((DIM+2)/(float)rc_block_size.x));

  dim3 block_size(CUDA_BLOCK_DIM,CUDA_BLOCK_DIM); /*Each cuda block is 2-d and has CUDA_BLOCK_DIM*CUDA_BLOCK_DIM threads*/
  dim3 grid_size((int)ceil(DIM/(float)CUDA_BLOCK_DIM),(int)ceil(DIM/(float)CUDA_BLOCK_DIM)); /*Number of cuda blocks in the grid*/
  
  hipMalloc((void **)&dev_array1,(DIM+2)*(DIM+2)*sizeof(char));
  hipMalloc((void **)&dev_array2,(DIM+2)*(DIM+2)*sizeof(char));

  hipMemcpy(dev_array1,h_array,(DIM+2)*(DIM+2)*sizeof(char),hipMemcpyHostToDevice);
  
  /*int* cell_sum_dev;
  int* flag_dev;
  int cell_sum = 0;
  int flag = 0;

  hipMalloc((void **)&cell_sum_dev,sizeof(int));
  hipMalloc((void **)&flag_dev,sizeof(int));*/

  GET_TIME(start); 

  /*Start Iterations*/
  for(i = 0; i < ITERS; i++){

    /*char file_name[100];
    sprintf(file_name, "./generations/%dGen.txt",i+1);*/
 
    /*Fill the extra perimeter of the previous generation array*/
    fill_rows<<<rows_grid_size,rc_block_size>>>(dev_array1);
    fill_columns<<<cols_grid_size,rc_block_size>>>(dev_array1);
    
    /*cell_sum = 0;
    flag = 0;
   
    hipMemcpy(cell_sum_dev,&cell_sum,sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(flag_dev,&flag,sizeof(int),hipMemcpyHostToDevice);*/

    /*Calculate the next game generation*/
    step<<<grid_size, block_size>>>(dev_array1,dev_array2/*,cell_sum_dev,flag_dev*/);

    /*Copy generation to cpu*/
    hipMemcpy(h_array,dev_array2,(DIM+2)*(DIM+2)*sizeof(char),hipMemcpyDeviceToHost);

    /*Copy reduction results to cpu*/
    /*hipMemcpy(&cell_sum,cell_sum_dev,sizeof(int),hipMemcpyDeviceToHost);
    hipMemcpy(&flag,flag_dev,sizeof(int),hipMemcpyDeviceToHost);*/

    /*Write the generation into a file*/
    /*int fd1 = open(file_name, O_RDWR | O_CREAT ,0666);
    if(fd1 < 0){
      fprintf(stderr, "Could not open file \"%s\"\n", file_name);
      return -1;
    }
  
    for(n = DIM+3;n < ((DIM+2)*(DIM+2))-(DIM+2);n+=DIM+2){
      write(fd1,&h_array[n],DIM);
      write(fd1,"\n",sizeof(char));
    }
    close(fd1);*/
    
    /*Terminate if grid is empty or hasnt changed*/
    /*if((cell_sum == 0) || (flag == 0)) break;*/

    /*Previous generation becomes next and vice versa*/
    temp_arr = dev_array1;
    dev_array1 = dev_array2;
    dev_array2 = temp_arr;
  }
  
  GET_TIME(finish);  

  printf("Elapsed time: %f seconds\n",finish-start);
  
  /*Free allocated resources at gpu and cpu*/
  hipFree(dev_array1);
  hipFree(dev_array2);
  free(h_array);

  return 0;
}  
  
  
